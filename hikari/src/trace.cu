#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstring>
#include <cassert>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "glad/glad.h"
#include "cuda_gl_interop.h"

#include "trace.hpp"

void cudaCheckErr() {
  // check for error
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}

void cudaAssert(hipError_t code, const char *file, int line, bool abort)
{
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);

    system("pause");

    if (abort) {
      exit(code);
    }
  }
}

CudaSurface::CudaSurface(unsigned int texid)
  : m_cudaGraphicsResource(nullptr)
  , m_cudaArray(nullptr)
  , m_mapped(false)
{
  // todo: make this short circuit if one fails
  cudaCheck(hipGraphicsGLRegisterImage(&m_cudaGraphicsResource, texid, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));
}

CudaSurface::~CudaSurface()
{
  if (m_mapped) {
    unmap();
  }
}


hipSurfaceObject_t CudaSurface::map() {
  assert(!m_mapped);
  if (!m_mapped) {
    m_mapped = true;

    cudaCheck(hipGraphicsMapResources(1, &m_cudaGraphicsResource));
    cudaCheck(hipGraphicsSubResourceGetMappedArray(&m_cudaArray, m_cudaGraphicsResource, 0, 0));

    hipResourceDesc pixelDescription;
    memset(&pixelDescription, 0, sizeof(pixelDescription));
    pixelDescription.resType = hipResourceTypeArray;
    pixelDescription.res.array.array = m_cudaArray;
    cudaCheck(hipCreateSurfaceObject(&m_pixelSurface, &pixelDescription));
  }

  return m_pixelSurface;
}

void CudaSurface::unmap() {
  assert(m_mapped);
  if (m_mapped) {
    m_mapped = false;

    cudaCheck(hipGraphicsUnmapResources(1, &m_cudaGraphicsResource));
    cudaCheck(hipDestroySurfaceObject(m_pixelSurface));
  }
}

__global__
void trace_kernel(hipSurfaceObject_t pixelSurface, float time) {
  int idx = blockIdx.x;
  int idy = blockIdx.y;
  surf2Dwrite<float4>(float4{fmod(time / 6.0f, 1.0f), 0.0f, 0.0f, 1.0f}, pixelSurface, idx * 16, idy);
}

int trace(CudaSurface* cudaSurface, float time) {
  auto surf = cudaSurface->map();
  trace_kernel<<<dim3(800, 800, 1), 1>>>(surf, time);
  cudaCheckErr();
  cudaSurface->unmap();
  return 0;
}